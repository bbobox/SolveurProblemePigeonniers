#include "hip/hip_runtime.h"
#include "my_cuda.h"
#include <cstdlib>
#include <iostream>
#include <stdlib.h>
#include <cstdint>
#include <cstring>
#include <time.h>
#include <math.h>

using namespace std;


int n;
int p ;

int nb_tot=0;


 /* calcul de temps d'execution */
void duree(time_t _begin, time_t _end)
{
  double temp;
  double hours=0, min=0, sec=0;
  double dureeCalc = difftime(_end, _begin);
  temp = modf(dureeCalc/3600., &hours);
  temp = modf(temp*60., &min);
  temp = modf(temp*60., &sec);
  std::cout<<"Duree du calcul : "<<hours<<" h "<<min<<" min "<<sec<<" sec"<<std::endl;
}




//__device__
void solution_initial(int* solution){
	for (int i=0;i<n*p;++i){

		solution[i]=0;
	}


}



__global__
void kernelsolveurPigeonniersBacktrack(int pigeon_id,int *solution,int n,int p){
	if (n>p){
		printf(" aucune solution  possible car la contrainte n<=p n'est pas satisfaite \n");
	}

	else{

		if(pigeon_id==n ){
			
		}
		else{
			if (threadIdx.y==pigeon_id && threadIdx.x<blockDim.x ){
				int idx=blockDim.x * threadIdx.y+ threadIdx.x;

				bool ok=true;
				int i=0;
				while (i<n && ok==true){
					if(solution[i*p+threadIdx.y]==1){
						ok=false;
					}
					i=i+1;
				}

				if (ok==true){
					int* solI=new int[n*p];
					for (int i=0;i<n*p;++i){

							solI[i]=0;
						}


					std::memcpy(solI,solution,(n*p)*sizeof(int));
					solI[idx]=1;


				 if (pigeon_id!=n){
					dim3 grid(1,1,1);
					dim3 block(p,n,1);
					printf(" n= %d et p= %d \n", n, p);
					//kernelsolveurPigeonniersBacktrack<<< grid,block >>> (pigeon_id+1,solI,n,p);
					//hipDeviceSynchronize();
				}


				}

			}
		}
	}
}



int main(int argc, char *argv[]){
	if (argv[1]!=NULL && argv[2]!=NULL){
		 	n=atoi(argv[1]);
			p=atoi(argv[2]);
		int* sol_cpu= new int[n*p];
		solution_initial(sol_cpu);

		int* sol_gpu;

		hipMalloc((void**)&sol_gpu,(n*p)*sizeof(int));

		hipMemcpy(sol_gpu,sol_cpu,(n*p)*sizeof(int),hipMemcpyHostToDevice);

		dim3 grid(1,1,1);
		dim3 block(p,n,1);

		kernelsolveurPigeonniersBacktrack<<< grid,block >>> (0,sol_gpu,n,p);
		hipDeviceSynchronize();

		hipFree(sol_gpu);


	}

return 0;
}

